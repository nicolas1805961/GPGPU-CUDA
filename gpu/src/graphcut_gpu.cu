#include "hip/hip_runtime.h"
#include "Image.hpp"
#include "Graph_gpu.hpp"
#include <iostream>
#include <fstream>
#include <vector>
#include <cmath>
#include <stdio.h>

#define cudaCheckError() {                                                                       \
  hipError_t e=hipGetLastError();                                                        \
  if(e!=hipSuccess) {                                                                     \
      printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));        \
      exit(EXIT_FAILURE);                                                                  \
  }                                                                                        \
}

__global__ void display(GraphGPU* graph)
{
    int x = blockIdx.x * 256 + threadIdx.x;

    printf("%d", graph->m_excessFlow[x]);
}

//function added to check if the graph was copied successfully on GPU
void count_active_cpu(GraphGPU graph)
{
    int count = 0;
    for (int i = 0; i < graph.m_maxHeight; i++)
    {
        if (graph.m_excessFlow[i] > 0 && graph.m_heights[i] < graph.m_maxHeight)
            count++;
    }
    std::cout << "cpu_count = " << count << "\n";
}

//Copy a cpu array onto the device
void copy_pointer(int** gpu_ptr, int* cpu_ptr, size_t size)
{
    //allocate array on the device
    int *host_ptr;
    hipMalloc((void **) &host_ptr, size);
    cudaCheckError();
    //copy the cpu array on the device
    hipMemcpy(host_ptr, cpu_ptr, size, hipMemcpyHostToDevice);
    cudaCheckError();
    //copy the gpu array adress on our pointer
    hipMemcpy(gpu_ptr, &host_ptr, sizeof(int*), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    cudaCheckError();
}

//Copy the CPU graph to the GPU
void copy_graph(GraphGPU *gpu_graph, GraphGPU cpu_graph)
{
    //copy the structure
    hipMemcpy(gpu_graph, &cpu_graph, sizeof(GraphGPU), hipMemcpyHostToDevice);
    cudaCheckError();

    size_t matrix_size = cpu_graph.m_maxHeight * sizeof(int);

    //copy every matrices used
    copy_pointer(&(gpu_graph->m_excessFlow), cpu_graph.m_excessFlow, matrix_size);
    copy_pointer(&(gpu_graph->m_heights), cpu_graph.m_heights, matrix_size);
    copy_pointer(&(gpu_graph->m_topNeighbourCapacity), 
        cpu_graph.m_topNeighbourCapacity, matrix_size);
    copy_pointer(&(gpu_graph->m_bottomNeighbourCapacity),
        cpu_graph.m_bottomNeighbourCapacity, matrix_size);
    copy_pointer(&(gpu_graph->m_rightNeighbourCapacity),
        cpu_graph.m_rightNeighbourCapacity, matrix_size);
    copy_pointer(&(gpu_graph->m_leftNeighbourCapacity),
        cpu_graph.m_leftNeighbourCapacity, matrix_size);
    copy_pointer(&(gpu_graph->m_sinkCapacity),
        cpu_graph.m_sinkCapacity, matrix_size);
    copy_pointer(&(gpu_graph->m_sourceCapacity),
        cpu_graph.m_sourceCapacity, matrix_size);
}

//copy the graph heights onto the swap
__global__ void graph_to_swap(GraphGPU *graph, int *swap)
{
    int x = blockIdx.x * 256 + threadIdx.x;
    if (x >= graph->m_maxHeight)
        return;

    swap[x] = graph->m_heights[x];
}

//copy the swap heights onto the graph heights
__global__ void swap_to_graph(GraphGPU *graph, int *swap)
{
    int x = blockIdx.x * 256 + threadIdx.x;
    if (x >= graph->m_maxHeight)
        return;

    graph->m_heights[x] = swap[x];
}


void graphcut_gpu(GraphGPU graph)
{
    hipDeviceProp_t device;
    hipGetDeviceProperties(&device,0);

    int blockSize = 256;
    int numBlocks = (graph.m_maxHeight + blockSize - 1) / blockSize;

    //allocate graph on GPU and copy the one initialize on CPU
    GraphGPU* g;
    hipMalloc((void **) &g, sizeof(GraphGPU));
    cudaCheckError();
    copy_graph(g, graph);

    //setup CPU counter
    int c = 0;
    int* count = &c;

    //setup GPU counter
    int* gpu_count;
    hipMalloc((void **)&gpu_count, sizeof(int));
    cudaCheckError();
    hipDeviceSynchronize();
    hipMemcpy(gpu_count, count, sizeof(int), hipMemcpyHostToDevice);
    cudaCheckError();
    hipDeviceSynchronize();

    //count active nodes
    count_active<<<numBlocks, blockSize>>>(g, gpu_count);
    cudaCheckError();
    hipDeviceSynchronize();

    //copy the counter from GPU to CPU
    hipMemcpy(count, gpu_count, sizeof(int), hipMemcpyDeviceToHost);

    //initialize swap heights
    int* swap_heights;
    hipMalloc((void **)&swap_heights, graph.m_maxHeight * sizeof(int));
    cudaCheckError();

    //copy the graph heights onto the swap
    graph_to_swap<<<numBlocks, blockSize>>>(g, swap_heights);

    while(*count > 0)
    {
        push<<<numBlocks, blockSize>>>(g);
        hipDeviceSynchronize();

        //update swap heights
        relabel<<<numBlocks, blockSize>>>(g, swap_heights);
        hipDeviceSynchronize();

        //copy the updated heights to the graph
        swap_to_graph<<<numBlocks, blockSize>>>(g, swap_heights);
        hipDeviceSynchronize();

        //set gpu counter to 0
        hipMemset(gpu_count,0,sizeof(int));
        count_active<<<numBlocks, blockSize>>>(g, gpu_count);
        cudaCheckError();
        hipDeviceSynchronize();

        //copy GPU counter onto CPU counter for the while loop
        hipMemcpy(count, gpu_count, sizeof(int), hipMemcpyDeviceToHost);
        std::cout << *count << "\n";
    }
}
