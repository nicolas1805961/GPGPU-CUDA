#include "hip/hip_runtime.h"
#include "Graph.hpp"
#include <limits>
#include <stdio.h>

__device__ int artificial_height = 10000;
//Same as CPU initialization
Graph::Graph(Image const& image, Image const& imageHelper)
{
    m_maxHeight = image.getWidth() * image.getHeight();
    m_width = image.getWidth();
    m_height = image.getHeight();

    m_excessFlow = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_heights = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_topNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_leftNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_rightNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_bottomNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_sourceCapacityToNodes = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_sinkCapacityToNodes = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_sourceCapacityFromNodes = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_sinkCapacityFromNodes = (int*)std::malloc(m_maxHeight * sizeof(int));

    m_grayImage = image.getImageGray();

    // Ici je label les arrêtes entre les noeuds avec la distance en rgb entre chaque pixel (noeud = pixel)
    for (int i = 0; i < m_height - 1; i++)
    {
        for (int j = 0; j < m_width; j++)
        {
            int distance = pow((m_grayImage[m_width * i + j] - m_grayImage[m_width * (i + 1) + j]), 2);
            m_bottomNeighbourCapacity[m_width * i + j] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
    }
    for (int i = 1; i < m_height; i++)
    {
        for (int j = 0; j < m_width; j++)
        {
            int distance = pow((m_grayImage[m_width * i + j] - m_grayImage[m_width * (i - 1) + j]), 2);
            m_topNeighbourCapacity[m_width * i + j] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
    }
    for (int i = 0; i < m_height; i++)
    {
        for (int j = 1; j < m_width; j++)
        {
            int distance = pow((m_grayImage[m_width * i + j] - m_grayImage[m_width * i + j - 1]), 2);
            m_leftNeighbourCapacity[m_width * i + j] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
        for (int j = 0; j < m_width - 1; j++)
        {
            int distance = pow((m_grayImage[m_width * i + j] - m_grayImage[m_width * i + j + 1]), 2);
            m_rightNeighbourCapacity[m_width * i + j] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
    }
    //contient les indices des pixels noirs et blanc
    std::vector<int> white;
    std::vector<int> black;
    int sumIntensityForegroundRed = 0;
    int sumIntensityForegroundGreen = 0;
    int sumIntensityForegroundBlue = 0;
    int sumIntensityBackgroundRed = 0;
    int sumIntensityBackgroundGreen = 0;
    int sumIntensityBackgroundBlue = 0;
    auto ptr = image.getImageRgb();
    auto ptrHelper = imageHelper.getImageRgb();
    // ici je label les arrêtes entre la source et les noeuds appartenant au foreground d'une part, et d'autre part entre les noeuds appartenant au background et le puit. Les noeuds dont on est sur qu'ils sont du foreground ou background sont les pixels blanc et noir respectivement. On les etiquettes avec une très grande valeur car c'est sur qu'ils appartiennent au background/foreground.
    for (int i = 0; i < imageHelper.getHeight() * imageHelper.getWidth(); i++)
    {
        if (ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 1] && ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 2] && ptrHelper[(i * 3)] == 255)
        {
            white.push_back(i * 3);
            m_sourceCapacityToNodes[i + i % m_width] = std::numeric_limits<int>::max(); //sqrt(pow(255, 2) * 3);
            m_sinkCapacityFromNodes[i + i % m_width] = 0;
        }
        else if (ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 1] && ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 2] && ptrHelper[(i * 3)] == 0)
        {
            black.push_back(i * 3);
            m_sinkCapacityFromNodes[i + i % m_width] = std::numeric_limits<int>::max(); //sqrt(pow(255, 2) * 3);
            m_sourceCapacityToNodes[i + i % m_width] = 0;
        }
    }

    for (size_t i = 0; i < white.size(); i++)
    {
        sumIntensityForegroundRed += ptr[white[i]];
        sumIntensityForegroundGreen += ptr[white[i] + 1];
        sumIntensityForegroundBlue += ptr[white[i] + 2];
    }
    for (size_t i = 0; i < black.size(); i++)
    {
        sumIntensityBackgroundRed += ptr[black[i]];
        sumIntensityBackgroundGreen += ptr[black[i] + 1];
        sumIntensityBackgroundBlue += ptr[black[i] + 2];
    }
    float averageForegroundRed = sumIntensityForegroundRed / white.size();
    float averageForegroundGreen = sumIntensityForegroundGreen / white.size();
    float averageForegroundBlue = sumIntensityForegroundBlue / white.size();
    float averageBackgroundRed = sumIntensityBackgroundRed / black.size();
    float averageBackgroundGreen = sumIntensityBackgroundGreen / black.size();
    float averageBackgroundBlue = sumIntensityBackgroundBlue / black.size();

    for (int i = 0; i < m_height; i++)
    {
        for (int j = 0; j < m_width; j++)
        {
            uint8_t r = image.getImageRgb()[m_width * (i * 3) + (j * 3)];
            uint8_t g = image.getImageRgb()[m_width * (i * 3) + (j * 3) + 1];
            uint8_t b = image.getImageRgb()[m_width * (i * 3) + (j * 3) + 2];

            float df = sqrt(pow(r - averageForegroundRed, 2) + pow(g - averageForegroundGreen, 2) + pow(b - averageForegroundBlue, 2));
            float db = sqrt(pow(r - averageBackgroundRed, 2) + pow(g - averageBackgroundGreen, 2) + pow(b - averageBackgroundBlue, 2));

            if (m_sourceCapacityToNodes[m_width * i + j] == 1)
                m_sourceCapacityToNodes[m_width * i + j] = ((df / sqrt(pow(255, 2) * 3)) - 1) * -sqrt(pow(255, 2) * 3); //-log(pf);;
            if (m_sinkCapacityFromNodes[m_width * i + j] == 1)
                m_sinkCapacityFromNodes[m_width * i + j] = ((db / sqrt(pow(255, 2) * 3)) - 1) * -sqrt(pow(255, 2) * 3); //-log(1 - pf);
        }
    }
    //ici j'initialise l'excess flow en saturant les arrêtes(capacity) partant de la source. Et je créer la capacité dans le sens inverse.
    for (int i = 0; i < m_height; i++)
    {
        for (int j = 0; j < m_width; j++)
        {
            m_excessFlow[m_width * i + j] = m_sourceCapacityToNodes[m_width * i + j] - m_sinkCapacityFromNodes[m_width * i + j];
            m_sourceCapacityFromNodes[m_width * i + j]= m_sourceCapacityToNodes[m_width * i + j];
            m_sourceCapacityToNodes[m_width * i + j] = 0;
        }
    }
}

//same as CPU push, added atomic operations to avoid random states
__global__ void push(Graph* graph)
{
    int x = blockIdx.x * 256 + threadIdx.x;
    int i = x / graph->m_width;
    int j = x % graph->m_width;

    int* m_excessFlow = graph->m_excessFlow;
    int* m_heights = graph->m_heights;
    int* m_leftNeighbourCapacity = graph->m_leftNeighbourCapacity;
    int* m_rightNeighbourCapacity = graph->m_rightNeighbourCapacity;
    int* m_topNeighbourCapacity = graph->m_topNeighbourCapacity;
    int* m_bottomNeighbourCapacity = graph->m_bottomNeighbourCapacity;
    int m_maxHeight = graph->m_maxHeight;
    int m_height = graph->m_height;
    int m_width = graph->m_width;

    if (x >= m_maxHeight)
        return;

    //artificially changed maxHeight to try to have a faster result
    m_maxHeight = artificial_height;

    if (m_excessFlow[i*m_width+j] > 0 && m_heights[i*m_width+j] < m_maxHeight)
    {
        if (j > 0 && m_heights[i*m_width + j - 1] == m_heights[i*m_width+j] - 1)
        {
            int flow = fminf(m_leftNeighbourCapacity[i*m_width+j], m_excessFlow[i*m_width+j]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[i*m_width+j],flow);
            //m_excessFlow[i*m_width+j - 1] += flow;
            atomicAdd(&m_excessFlow[i*m_width+j - 1],flow);
            //m_leftNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_leftNeighbourCapacity[i*m_width+j], flow);
            //m_rightNeighbourCapacity[i*m_width+j - 1] += flow;
            atomicAdd(&m_rightNeighbourCapacity[i*m_width+j-1],flow);
        }
        if (j < m_width - 1 && m_heights[i*m_width+j + 1] == m_heights[i*m_width+j] - 1)
        {
            int flow = fminf(m_rightNeighbourCapacity[i*m_width+j], m_excessFlow[i*m_width+j]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[i*m_width+j],flow);
            //m_excessFlow[i*m_width+j + 1] += flow;
            atomicAdd(&m_excessFlow[i*m_width+j + 1],flow);
            //m_rightNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_rightNeighbourCapacity[i*m_width+j], flow);
            //m_leftNeighbourCapacity[i*m_width+j + 1] += flow;
            atomicAdd(&m_leftNeighbourCapacity[i*m_width+j+1],flow);
        }
        if (i > 0 && m_heights[(i - 1)*m_width+j] == m_heights[i*m_width+j] - 1)
        {
            int flow = fminf(m_topNeighbourCapacity[i*m_width+j], m_excessFlow[i*m_width+j]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[i*m_width+j],flow);
            //m_excessFlow[(i-1)*m_width+j] += flow;
            atomicAdd(&m_excessFlow[(i-1)*m_width+j],flow);
            //m_topNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_topNeighbourCapacity[i*m_width+j], flow);
            //m_bottomNeighbourCapacity[(i-1)*m_width+j] += flow;
            atomicAdd(&m_bottomNeighbourCapacity[(i-1)*m_width+j],flow);
        }
        if (i < m_height - 1 && m_heights[(i + 1)*m_width+j] == m_heights[i*m_width+j] - 1)
        {
            int flow = fminf(m_bottomNeighbourCapacity[i*m_width+j], m_excessFlow[i*m_width+j]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[i*m_width+j],flow);
            //m_excessFlow[(i+1)*m_width + j] += flow;
            atomicAdd(&m_excessFlow[(i+1)*m_width+j],flow);
            //m_bottomNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_bottomNeighbourCapacity[i*m_width+j], flow);
            //m_topNeighbourCapacity[(i + 1)*m_width+j] += flow;
            atomicAdd(&m_topNeighbourCapacity[(i+1)*m_width+j],flow);
        }
    }
    //printf("i=%d j=%d\n",i, j);
}

//Same as CPU relabel, but pushing values on swap and reading from actual heights
__global__ void relabel(Graph* graph, int* swap_heights)
{
    int x = blockIdx.x * 256 + threadIdx.x;
    int i = x / graph->m_width;
    int j = x % graph->m_width;

    int* m_excessFlow = graph->m_excessFlow;
    int* m_heights = graph->m_heights;
    int* m_leftNeighbourCapacity = graph->m_leftNeighbourCapacity;
    int* m_rightNeighbourCapacity = graph->m_rightNeighbourCapacity;
    int* m_topNeighbourCapacity = graph->m_topNeighbourCapacity;
    int* m_bottomNeighbourCapacity = graph->m_bottomNeighbourCapacity;
    int m_maxHeight = graph->m_maxHeight;
    int m_width = graph->m_width;

    if (x >= m_maxHeight)
        return;

    m_maxHeight = artificial_height;


    if (m_excessFlow[i*m_width + j] > 0 && m_heights[i*m_width+j] < m_maxHeight)
    {
        auto myHeight = m_maxHeight;
        /*if (m_sinkCapacityFromNodes[i][j] > 0)
            myHeight = std::min(myHeight, 0);*/
        if (m_leftNeighbourCapacity[i*m_width+j] > 0)
            myHeight = min(myHeight, m_heights[i*m_width + j - 1] + 1);
        if (m_rightNeighbourCapacity[i*m_width+j] > 0)
            myHeight = min(myHeight, m_heights[i*m_width + j + 1] + 1);
        if (m_topNeighbourCapacity[i*m_width+j] > 0)
            myHeight = min(myHeight, m_heights[(i-1)*m_width + j] + 1);
        if (m_bottomNeighbourCapacity[i*m_width+j] > 0)
            myHeight = min(myHeight, m_heights[(i+1)*m_width+j] + 1);
        /*if (m_sourceCapacityFromNodes[i][j] > 0)
            myHeight = std::min(myHeight, m_maxHeight);*/
        swap_heights[i*m_width+j] = myHeight;
    }
}

//count how many nodes are active
__global__ void count_active(Graph* graph, int* count)
{
    int x = blockIdx.x * 256 + threadIdx.x;
    /*int i = x / graph->m_width;
    int j = x % graph->m_height;*/

    int m_maxHeight = graph->m_maxHeight;

    if (x >= m_maxHeight)
        return;

    m_maxHeight = artificial_height;

    if (graph->m_excessFlow[x] > 0  && graph->m_heights[x] < m_maxHeight)
    {
        atomicAdd(count, 1);
    }
}

