#include "hip/hip_runtime.h"
#include "Graph.hpp"
#include <limits>
#include <stdio.h>

__device__ int artificial_height = 5;
//Same as CPU initialization
Graph::Graph(Image const& image, Image const& imageHelper)
{
    m_maxHeight = image.getWidth() * image.getHeight();
    m_width = image.getWidth();
    m_height = image.getHeight();

    m_excessFlow = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_heights = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_topNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_leftNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_rightNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_bottomNeighbourCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_sourceCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));
    m_sinkCapacity = (int*)std::malloc(m_maxHeight * sizeof(int));

    for (int i = 0; i < m_maxHeight; i++)
    {
        m_excessFlow[i] = 0;
        m_heights[i] = 0;
        m_topNeighbourCapacity[i] = 0;
        m_leftNeighbourCapacity[i] = 0;
        m_rightNeighbourCapacity[i] = 0;
        m_bottomNeighbourCapacity[i] = 0;
        m_sourceCapacity[i] = 1;
        m_sinkCapacity[i] = 1;
    }
    
    m_grayImage = image.getImageGray();

    // Ici je label les arrêtes entre les noeuds avec la distance en rgb entre chaque pixel (noeud = pixel)
    for (int i = 0; i < m_height * m_width; i++)
    {
        if (i / m_width < m_height - 1)
        {
            int distance = pow((m_grayImage[i] - m_grayImage[i + m_width]), 2);
            m_bottomNeighbourCapacity[i] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
        if (i / m_width > 0)
        {
            int distance = pow((m_grayImage[i] - m_grayImage[i - m_width]), 2);
            m_topNeighbourCapacity[i] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
        if (i % m_width > 0)
        {
            int distance = pow((m_grayImage[i] - m_grayImage[i - 1]), 2);
            m_leftNeighbourCapacity[i] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
        if (i % m_width != m_width - 1)
        {
            int distance = pow((m_grayImage[i] - m_grayImage[i + 1]), 2);
            m_rightNeighbourCapacity[i] = ((sqrt(distance) / sqrt(pow(255, 2))) - 1) * -sqrt(pow(255, 2));
        }
    }
    //contient les indices des pixels noirs et blanc
    std::vector<int> white;
    std::vector<int> black;
    int sumIntensityForegroundRed = 0;
    int sumIntensityForegroundGreen = 0;
    int sumIntensityForegroundBlue = 0;
    int sumIntensityBackgroundRed = 0;
    int sumIntensityBackgroundGreen = 0;
    int sumIntensityBackgroundBlue = 0;
    auto ptr = image.getImageRgb();
    auto ptrHelper = imageHelper.getImageRgb();
    // ici je label les arrêtes entre la source et les noeuds appartenant au foreground d'une part, et d'autre part entre les noeuds appartenant au background et le puit. Les noeuds dont on est sur qu'ils sont du foreground ou background sont les pixels blanc et noir respectivement. On les etiquettes avec une très grande valeur car c'est sur qu'ils appartiennent au background/foreground.
    for (int i = 0; i < imageHelper.getHeight() * imageHelper.getWidth(); i++)
    {
        if (ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 1] && ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 2] && ptrHelper[(i * 3)] == 255)
        {
            white.push_back(i * 3);
            m_sourceCapacity[i] = std::numeric_limits<int>::max(); //sqrt(pow(255, 2) * 3);
            m_sinkCapacity[i] = 0;
        }
        else if (ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 1] && ptrHelper[(i * 3)] == ptrHelper[(i * 3) + 2] && ptrHelper[(i * 3)] == 0)
        {
            black.push_back(i * 3);
            m_sinkCapacity[i] = std::numeric_limits<int>::max(); //sqrt(pow(255, 2) * 3);
            m_sourceCapacity[i] = 0;
        }
    }

    for (size_t i = 0; i < white.size(); i++)
    {
        sumIntensityForegroundRed += ptr[white[i]];
        sumIntensityForegroundGreen += ptr[white[i] + 1];
        sumIntensityForegroundBlue += ptr[white[i] + 2];
    }
    for (size_t i = 0; i < black.size(); i++)
    {
        sumIntensityBackgroundRed += ptr[black[i]];
        sumIntensityBackgroundGreen += ptr[black[i] + 1];
        sumIntensityBackgroundBlue += ptr[black[i] + 2];
    }
    float averageForegroundRed = sumIntensityForegroundRed / white.size();
    float averageForegroundGreen = sumIntensityForegroundGreen / white.size();
    float averageForegroundBlue = sumIntensityForegroundBlue / white.size();
    float averageBackgroundRed = sumIntensityBackgroundRed / black.size();
    float averageBackgroundGreen = sumIntensityBackgroundGreen / black.size();
    float averageBackgroundBlue = sumIntensityBackgroundBlue / black.size();

    for (int i = 0; i < m_height * m_width; i++)
    {
        uint8_t r = image.getImageRgb()[i * 3];
        uint8_t g = image.getImageRgb()[i * 3 + 1];
        uint8_t b = image.getImageRgb()[i * 3 + 2];

        float df = sqrt(pow(r - averageForegroundRed, 2) + pow(g - averageForegroundGreen, 2) + pow(b - averageForegroundBlue, 2));
        float db = sqrt(pow(r - averageBackgroundRed, 2) + pow(g - averageBackgroundGreen, 2) + pow(b - averageBackgroundBlue, 2));

        if (m_sourceCapacity[i] == 1)
            m_sourceCapacity[i] = ((df / sqrt(pow(255, 2) * 3)) - 1) * -sqrt(pow(255, 2) * 3); //-log(pf);;
        if (m_sinkCapacity[i] == 1)
            m_sinkCapacity[i] = ((db / sqrt(pow(255, 2) * 3)) - 1) * -sqrt(pow(255, 2) * 3); //-log(1 - pf);
    }
    //ici j'initialise l'excess flow en saturant les arrêtes(capacity) partant de la source. Et je créer la capacité dans le sens inverse.
    for (int i = 0; i < m_height * m_width; i++)
    {
        m_excessFlow[i] = m_sourceCapacity[i] - m_sinkCapacity[i];
    }
}

//same as CPU push, added atomic operations to avoid random states
__global__ void push(Graph* graph)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    int* m_excessFlow = graph->m_excessFlow;
    int* m_heights = graph->m_heights;
    int* m_leftNeighbourCapacity = graph->m_leftNeighbourCapacity;
    int* m_rightNeighbourCapacity = graph->m_rightNeighbourCapacity;
    int* m_topNeighbourCapacity = graph->m_topNeighbourCapacity;
    int* m_bottomNeighbourCapacity = graph->m_bottomNeighbourCapacity;
    int m_maxHeight = graph->m_maxHeight;
    int m_height = graph->m_height;
    int m_width = graph->m_width;

    if (x >= m_maxHeight)
        return;

    //artificially changed maxHeight to try to have a faster result
    m_maxHeight = artificial_height;

    if (m_excessFlow[x] > 0 && m_heights[x] < m_maxHeight)
    {
        if (x % m_width > 0 && m_heights[x - 1] == m_heights[x] - 1)
        {
            int flow = fminf(m_leftNeighbourCapacity[x], m_excessFlow[x]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[x],flow);
            //m_excessFlow[i*m_width+j - 1] += flow;
            atomicAdd(&m_excessFlow[x - 1],flow);
            //m_leftNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_leftNeighbourCapacity[x], flow);
            //m_rightNeighbourCapacity[i*m_width+j - 1] += flow;
            atomicAdd(&m_rightNeighbourCapacity[x-1],flow);
        }
        if (x % m_width != m_width - 1 && m_heights[x + 1] == m_heights[x] - 1)
        {
            int flow = fminf(m_rightNeighbourCapacity[x], m_excessFlow[x]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[x],flow);
            //m_excessFlow[i*m_width+j + 1] += flow;
            atomicAdd(&m_excessFlow[x + 1],flow);
            //m_rightNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_rightNeighbourCapacity[x], flow);
            //m_leftNeighbourCapacity[i*m_width+j + 1] += flow;
            atomicAdd(&m_leftNeighbourCapacity[x+1],flow);
        }
        if (x / m_width > 0 && m_heights[x - m_width] == m_heights[x] - 1)
        {
            int flow = fminf(m_topNeighbourCapacity[x], m_excessFlow[x]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[x],flow);
            //m_excessFlow[(i-1)*m_width+j] += flow;
            atomicAdd(&m_excessFlow[x - m_width],flow);
            //m_topNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_topNeighbourCapacity[x], flow);
            //m_bottomNeighbourCapacity[(i-1)*m_width+j] += flow;
            atomicAdd(&m_bottomNeighbourCapacity[x - m_width],flow);
        }
        if (x / m_width < m_height - 1 && m_heights[x + m_width] == m_heights[x] - 1)
        {
            int flow = fminf(m_bottomNeighbourCapacity[x], m_excessFlow[x]);
            //m_excessFlow[i*m_width+j] -= flow;
            atomicSub(&m_excessFlow[x],flow);
            //m_excessFlow[(i+1)*m_width + j] += flow;
            atomicAdd(&m_excessFlow[x + m_width],flow);
            //m_bottomNeighbourCapacity[i*m_width+j] -= flow;
            atomicSub(&m_bottomNeighbourCapacity[x], flow);
            //m_topNeighbourCapacity[(i + 1)*m_width+j] += flow;
            atomicAdd(&m_topNeighbourCapacity[x + m_width],flow);
        }
    } 
    //printf("i=%d j=%d\n",i, j);
}

//Same as CPU relabel, but pushing values on swap and reading from actual heights
__global__ void relabel(Graph* graph, int* swap_heights)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    int* m_excessFlow = graph->m_excessFlow;
    int* m_heights = graph->m_heights;
    int* m_leftNeighbourCapacity = graph->m_leftNeighbourCapacity;
    int* m_rightNeighbourCapacity = graph->m_rightNeighbourCapacity;
    int* m_topNeighbourCapacity = graph->m_topNeighbourCapacity;
    int* m_bottomNeighbourCapacity = graph->m_bottomNeighbourCapacity;
    int m_maxHeight = graph->m_maxHeight;
    int m_width = graph->m_width;

    if (x >= m_maxHeight)
        return;

    m_maxHeight = artificial_height;


    if (m_excessFlow[x] > 0 && m_heights[x] < m_maxHeight)
    {
        auto myHeight = m_maxHeight;
        if (m_leftNeighbourCapacity[x] > 0)
            myHeight = min(myHeight, m_heights[x - 1] + 1);
        if (m_rightNeighbourCapacity[x] > 0)
            myHeight = min(myHeight, m_heights[x + 1] + 1);
        if (m_topNeighbourCapacity[x] > 0)
            myHeight = min(myHeight, m_heights[x - m_width] + 1);
        if (m_bottomNeighbourCapacity[x] > 0)
            myHeight = min(myHeight, m_heights[x + m_width] + 1);
        swap_heights[x] = myHeight;
    }
}

//count how many nodes are active
__global__ void count_active(Graph* graph, int* count)
{
    int x = blockIdx.x * 256 + threadIdx.x;
    /*int i = x / graph->m_width;
    int j = x % graph->m_height;*/

    int m_maxHeight = graph->m_maxHeight;

    if (x >= m_maxHeight)
        return;

    m_maxHeight = artificial_height;

    if (graph->m_excessFlow[x] > 0  && graph->m_heights[x] < m_maxHeight)
    {
        atomicAdd(count, 1);
    }
}

